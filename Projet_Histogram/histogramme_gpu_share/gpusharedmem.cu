#include "hip/hip_runtime.h"
#include "util.h"

#define SIZE (100*1024*1024)

__global__ void histo_kernel(unsigned char *buffer,long size, unsigned int *histo){
	
	__shared__ unsigned int temp[256];
	temp[threadIdx.x]=0;
	int i = threadIdx.x + blockIdx.x *blockDim.x;
	int offset = blockDim.x *gridDim.x;
	while(i<size){
		atomicAdd(&(histo[buffer[i]]),1);
		i+=offset;
	}
	
	__syncthreads();
	atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );

}



int main(int argc, char *argv[]){

//	unsigned char *buffer = (unsigned char *) big_random_block(SIZE);
	
	
	
	        if(argc <= 2){
        fprintf(stderr, "Arguments non valide");
        return 1;
        }
        FILE *f_input;
        FILE *f_output;

        long lSize;
        char *buffer;

        f_input = fopen ( argv[1] , "r" );
        f_output = fopen( argv[2],"w");
        if( !f_input ) perror(argv[1]),exit(1);


        fseek( f_input , 0L , SEEK_END);
        lSize = ftell( f_input );
        rewind( f_input );

        printf("The size is : %li", lSize);

        //buffer = calloc( 1, lSize+1 );
        buffer =(char*) malloc(lSize);
        if( !buffer ) fclose(f_input),fputs("memory alloc fails",stderr),exit(1);


       if( 1!=fread( buffer , lSize, 1 , f_input) )
          fclose(f_input),free(buffer),fputs("entire read fails",stderr),exit(1);	
	
	
	
	
	
	
	hipEvent_t start, stop;
	HANDLE_ERROR( hipEventCreate( &start ));
	HANDLE_ERROR( hipEventCreate( &stop ));    
	HANDLE_ERROR( hipEventRecord( start, 0));

	unsigned char *dev_buffer;
	unsigned int *dev_histo;

	HANDLE_ERROR( hipMalloc( (void**)&dev_buffer, lSize));
	HANDLE_ERROR( hipMemcpy( dev_buffer, buffer, lSize, hipMemcpyHostToDevice ) );    
	HANDLE_ERROR( hipMalloc( (void**)&dev_histo, 256 * sizeof( long )));    
	HANDLE_ERROR( hipMemset( dev_histo, 0, 256 * sizeof( int )));

	hipDeviceProp_t  prop;    
	HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
	int blocks = prop.multiProcessorCount;    
	histo_kernel<<<blocks*2,256>>>( dev_buffer, lSize, dev_histo );



	unsigned int histo[256];    
	HANDLE_ERROR( hipMemcpy( histo, dev_histo,256 * sizeof( int ),hipMemcpyDeviceToHost));
	for(int i =32;i< 128;i++){
            printf("%c:%d\n",i,histo[i]);
            fprintf(f_output, "%c:%d\n",i,histo[i]);
        }

	HANDLE_ERROR( hipEventRecord( stop, 0 ) );    
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	float   elapsedTime;    
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );    
	printf( "Time to generate:  %3.1f ms\n", elapsedTime );
		

	HANDLE_ERROR( hipEventDestroy( start ) );    
	HANDLE_ERROR( hipEventDestroy( stop ) );    
	
	hipFree( dev_histo );    
	hipFree( dev_buffer );    
	fclose(f_input);
	fclose(f_output);
	free( buffer );
	return 0; 


	}



