
#include <hip/hip_runtime.h>
#include <stdio.h>

#define START 32 
#define END 126
#define NBR 68 

__global__ void histo_kernel(unsigned char *buffer,long size, unsigned int *histo){

	int dt = 32;
	int i = threadIdx.x + blockIdx.x *blockDim.x;
	int stride = blockDim.x *gridDim.x;
	while(i<size){
		
		if (buffer[i] >= 32 && buffer[i] < 97)
                
            		atomicAdd(&(histo[buffer[i]-dt]),1);	
		if (buffer[i] >=97 && buffer[i] <= 122)
                	atomicAdd(&(histo[buffer[i]-dt-32]),1);
	
            	if (buffer[i] > 122 && buffer[i] <= 127 )
                	atomicAdd(&(histo[buffer[i]-dt -32 -26]),1);
			
		
			i+=stride;
	}

}



int main(int argc, char *argv[]){


	if(argc <= 2){
	fprintf(stderr, "Arguments non valide");
	return 1;
	}
	FILE *f_input;
	FILE *f_output;
	
	long lSize;
	char *buffer;

	f_input = fopen ( argv[1] , "r" );
	f_output = fopen( argv[2],"w");
        if( !f_input ) perror(argv[1]),exit(1);
	

        fseek( f_input , 0L , SEEK_END);
        lSize = ftell( f_input );
        rewind( f_input );

	printf("The size is : %li", lSize);
	    
        //buffer = calloc( 1, lSize+1 );
       	buffer =(char*) malloc(lSize);
        if( !buffer ) fclose(f_input),fputs("memory alloc fails",stderr),exit(1);

        
       if( 1!=fread( buffer , lSize, 1 , f_input) )
          fclose(f_input),free(buffer),fputs("entire read fails",stderr),exit(1);
		
	
	hipEvent_t start, stop;
 	hipEventCreate( &start );
	hipEventCreate( &stop );    
	hipEventRecord( start, 0);

	unsigned char *dev_buffer;
	unsigned int *dev_histo;

	 hipMalloc( (void**)&dev_buffer, lSize);
	 hipMemcpy( dev_buffer, buffer, lSize, hipMemcpyHostToDevice );    
	 hipMalloc( (void**)&dev_histo, 256 * sizeof( long ));    
	 hipMemset( dev_histo, 0, 256 * sizeof( int ));

	hipDeviceProp_t  prop;    
	hipGetDeviceProperties( &prop, 0  );
	int multiproc = prop.multiProcessorCount;    
        dim3  blocks(multiproc*2,1,1);
        dim3  threads(NBR, 1, 1);

	histo_kernel<<<blocks,threads>>>( dev_buffer, lSize, dev_histo );



	unsigned int histo[NBR];    

	hipMemcpy( histo, dev_histo,NBR * sizeof( int ),hipMemcpyDeviceToHost);
	int dt =32;
	for(int i =0;i< 68;i++){
		
        if(i>=0 && i<= 31&& (i+dt != 42) && (i+dt != 36)){
            printf("%c:%d\n",i+dt,histo[i]);
        }

        if(i>31 && i<= 57 ){
            printf("%c:%d\n",i+dt+32,histo[i]);
        }

        if(i> 57 && i <=64)
            printf("%c:%d\n",i+dt,histo[i]);

        if(i>64)
            printf("%c:%d\n",i+dt+26,histo[i]);
	
	}
	hipEventRecord( stop, 0 ) ;    
	hipEventSynchronize( stop );
	float   elapsedTime;    
	hipEventElapsedTime( &elapsedTime, start, stop  );    
	printf( "Time to generate:  %3.1f ms\n", elapsedTime );


	hipEventDestroy( start ) ;    
	hipEventDestroy( stop );    


	/*Free space*/
	hipFree( dev_histo );    
	hipFree( dev_buffer );
	fclose(f_input);
	fclose(f_output);
	free(buffer);
	return 0; 


}
