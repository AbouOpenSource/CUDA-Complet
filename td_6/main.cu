#include "hip/hip_runtime.h"
#include "stdio.h"
#define THREADS_PER_BLOCK 512
#define N (2048*2048)
__global__ void add(int *a, int *b, int *c){

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < N)
	c[index] = a[index] + b[index];
	printf('index: %d',threadIdx.x);
}
int main(void ){

	int *a,*b,*c; // host copies of a, b and c
	int *dev_a,*dev_b, *dev_c; // device copies of a, b and c
	int size = N * sizeof(int); // we need space for an integer


	//allocate device copies of a, b , c
	hipMalloc((void**) &dev_a, size);
	hipMalloc((void**) &dev_b, size);
	hipMalloc((void**) &dev_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	//random_ints(a,N);
	//random_ints(b,N);
	for (int i= 0; i<N ; i++){
		a[i]=i;
		b[i]=i*2;
		}

	//copy inputs to device (GPU)
	hipMemcpy(dev_a, a, size , hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	
	// launch add() kernel on GPU, passing parameters
	add<<<N/THREADS_PER_BLOCK  , THREADS_PER_BLOCK >>> (dev_a,dev_b,dev_c);
	
	//copy device result back to host copy of c 
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	/*for(int i =0; i<N; i++){
		printf("The value of the %d plus %d is : %d\n", a[i], b[i], c[i]);
		}*/


	free(a);
	free(b);
	free(c);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
		
	return 0;
}
