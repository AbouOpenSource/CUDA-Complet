
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NBR 68
__global__ void histo_kernel(unsigned char *buffer,long size, unsigned int *histo){
	
	 __shared__ unsigned int temp[68];
        int dt = 32;
	temp[threadIdx.x]=0;
        int i = threadIdx.x + blockIdx.x *blockDim.x;
        int offset = blockDim.x *gridDim.x;
        while(i<size){
                
            if (buffer[i] >= 32 && buffer[i] < 97)
                   //  histo[buffer[i]-dt]++;
            	atomicAdd(&temp[buffer[i]-dt],1);
	    if (buffer[i] >=97 && buffer[i] <= 122)
                atomicAdd(&temp[buffer[i] -dt -32],1);
		   // histo[buffer[i] - dt - 32]++;
            if (buffer[i] > 122 && buffer[i] <= 127 )
                   // histo[buffer[i] - dt - 32 - 26]++;
		atomicAdd(&temp[buffer[i]-dt -32-26],1);
                   i+=offset;
        }

        __syncthreads();
        atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );


}



int main(int argc, char *argv[]){

//	unsigned char *buffer = (unsigned char *) big_random_block(SIZE);
	
	
	
	if(argc <= 2){
        	fprintf(stderr, "Arguments non valide");
        	return 1;
        }
        /*For file input file and output file*/
	FILE *f_input;
        FILE *f_output;
	/*Will content the number of char in the input file*/
        long lSize;
        /*will content the file in char format*/
	char *buffer;
	/*Open the */
        f_input = fopen ( argv[1] ,"r" );
        f_output = fopen( argv[2],"w");


        if( !f_input ) perror(argv[1]),exit(1);


        fseek( f_input , 0L , SEEK_END);
        lSize = ftell( f_input );
        rewind( f_input );

       

        //buffer = calloc( 1, lSize+1 );
        buffer =(char*) malloc(lSize);
        if( !buffer ) fclose(f_input),fputs("memory alloc fails",stderr),exit(1);


       if( 1!=fread( buffer , lSize, 1 , f_input) )
          fclose(f_input),free(buffer),fputs("entire read fails",stderr),exit(1);	
	
	
	
	
	
	/*Create event for co;pute running time*/
	hipEvent_t start, stop;
        hipEventCreate( &start );
	hipEventCreate( &stop );    
        /*Launch event to specify the start of running*/
	hipEventRecord( start, 0);


	/*allocate device memory*/
	unsigned char *dev_buffer;
	unsigned int *dev_histo;
	/*Give space in Global memory of GPU to store different variable*/
	hipMalloc( (void**)&dev_buffer, lSize);
	/*Copy from CPU Global memory to GPU Global memory*/
	hipMemcpy( dev_buffer, buffer, lSize, hipMemcpyHostToDevice  );    
	/*Create space for histo variable and initialize at 0 each slopt*/
	hipMalloc( (void**)&dev_histo, NBR * sizeof( long));    
	hipMemset( dev_histo, 0, NBR * sizeof( int ));    

	/*Define of the configuration for kernel running*/
 	hipDeviceProp_t  proprieties;
        hipGetDeviceProperties( &proprieties, 0  );
        int multiproc = proprieties.multiProcessorCount;
        dim3  blocks(multiproc*2,1,1);
        dim3  threads(NBR, 1, 1);



	histo_kernel<<<blocks,threads>>>( dev_buffer, lSize, dev_histo );


	/*Define histo vqriqble and copy on GPU global memory*/
	unsigned int histo[NBR];    
	hipMemcpy( histo, dev_histo,NBR * sizeof( int ),hipMemcpyDeviceToHost);
	int dt =32;	
	for(int i =0;i< NBR;i++){

        if((i>=0 && i<= 31 && (i+dt != 42) && (i+dt != 36)) || (i>58 && i<=64) )
            fprintf(f_output, "%c:%d\n",i+dt,histo[i]);

        if(i>31 && i<= 58 )
	    fprintf(f_output, "%c:%d\n",i+dt+32,histo[i]);
	
       // if(i> 58 && i <=64)
         //   fprintf(f_output, "%c:%d\n",i+dt,histo[i]);	
        
	if(i>64)
            fprintf(f_output, "%c:%d\n",i+dt+26,histo[i]);
 

	}
	
	/*Get event at the end of loop*/
	hipEventRecord( stop, 0  );    
	hipEventSynchronize( stop );
	float   elapsedTime;    
        hipEventElapsedTime( &elapsedTime, start, stop );    
	printf( "Time of running :  %3.1f ms\n", elapsedTime );
		
	/*Destroy event for running time*/
	hipEventDestroy( start );    
	hipEventDestroy( stop );    
	
	
	/*Free memory and close the files**/
	hipFree( dev_histo );    
	hipFree( dev_buffer );    
	fclose(f_input);
	fclose(f_output);
	free( buffer );
	return 0; 


	}



