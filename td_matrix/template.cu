#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(int *refmat, int *idata, const unsigned int i_dim, const unsigned int j_dim);
////////////////////////////////////////////////////////////////////////////////
//! Simple transpose kernel
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transpose_share(int *g_idata, int *g_odata)
{



	__shared__ float tile[4096];

  	int x = blockIdx.x * blockDim.x + threadIdx.x;
  	int y = blockIdx.y * blockDim.y + threadIdx.y;

  	int width = gridDim.x * blockDim.x;
  	int height = gridDim.y * blockDim.y;
  	// Copy global memory in shared memory
  	for (int j =blockIdx.x*4096; j < 4096*4096; j+=1)
     		tile[ j-blockIdx.x*4096 ] = g_idata[(y+j)*4096 + x];

  	__syncthreads();
  	//Copy from shared memory to global memory 	
  	x = blockIdx.y * blockDim.y + threadIdx.y; 
  	y = blockIdx.x * blockDim.x + threadIdx.x;

  	for (int j = blockIdx.x*4096; j <4096*4096; j +=1)
    		g_odata[(y+j)*width + x] = tile[j-blockIdx.x*4096];

	 
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    const unsigned int I_DIM = 4096;
    const unsigned int J_DIM = 4096;
    unsigned int mem_size = sizeof(int) * I_DIM * J_DIM;

    // allocate host memory
    int *h_idata = (int *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < I_DIM; i++)
      for (unsigned int j = 0; j < J_DIM; j++)
      {
//	printf("Index est : %d egale a %d\n",i*J_DIM+j,j);
      	      h_idata[i*J_DIM +j] = i;
      }

    // allocate device memory
    int *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    int *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    const unsigned int BSX = 16;
    const unsigned int BSY = 16;
    dim3  blocks(BSX, BSY, 1);
    dim3  grid(J_DIM/BSX, I_DIM/BSY, 1);

    // execute the kernel
    transpose_share<<< grid, blocks, 0 >>>(d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    int *h_odata = (int *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution
    int *reference = (int *) malloc(mem_size);
    computeGold(reference, h_idata, I_DIM, J_DIM);

    // check result
    bool resultOk = true;
    for (unsigned int i = 0; i < I_DIM; i++)
      for (unsigned int j = 0; j < J_DIM; j++)
      {
        if (reference[i*J_DIM +j] != h_odata[i*J_DIM +j]){
          resultOk = false;
          break;
        }
      }
    if(resultOk)
      printf("TEST PASSED\n");
    else
      printf("TEST FAILED\n");


    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
