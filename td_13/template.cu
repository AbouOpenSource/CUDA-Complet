#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
typedef float DTYPE;
extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
computeNstore(DTYPE *out_data, unsigned int nsteps)
{
	const unsigned int tidb = threadIdx.x;
	const unsigned int tid  = blockIdx.x * blockDim.x +tidb;

	DTYPE step ;
	if(tid < nsteps ){
		step = (1.0)/((DTYPE)nsteps);
		DTYPE x = ((DTYPE) tid +0.5) * step ;
		out_data[tid] = (DTYPE) 1.0 / (1.0 + x*x);
	}
}
__global__ void summup(DTYPE *data,unsigned int nvalues){

	const unsigned int tidb = threadIdx.x;
        const unsigned int tid = blockIdx.x * blockDim.x + tidb;

	extern __shared__ DTYPE sdata[];

	//mandatory if nvalues is note a power of 2
       if(tid< nvalues)
       	sdata[tidb] = data[tid];
       else
	sdata[tidb] = 0.0;

 	__syncthreads();

  	// loop until all values have been summed up
	for(int stride =1; stride < blockDim.x; stride *=2 ){
		if(tidb %(2*stride) == 0)
		  	sdata[tidb] += sdata[tidb + stride];
		__syncthreads();
	}	
	//writes the sum of the bloc 
	if(tidb == 0 )
		data[blockIdx.x] = sdata[tidb];
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
//    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);


    unsigned int threads_per_block = 1024;
    unsigned int nsteps = atoi(argv[1]);
    unsigned int mem_size = sizeof(DTYPE) * nsteps;

    DTYPE *d_vector;
    checkCudaErrors(hipMalloc((void **) &d_vector,mem_size)); 
	
    // setup execution parameter
    dim3  grid((nsteps+threads_per_block-1)/threads_per_block, 1, 1);
    dim3  threads(threads_per_block, 1, 1);
    printf("juste avant la functio");
    // execute the kernel
    	computeNstore<<< grid, threads, 0 >>>(d_vector, nsteps);
	printf("computeNstore terminated \n");
	int nvalues = nsteps;
	int nblocks = (nvalues+threads_per_block-1)/threads_per_block;
	
	while(nvalues >1){
	printf("vector : %d values - %d block(s) of %d threads \n",nvalues,nblocks,threads_per_block);
	summup<<<nblocks, threads_per_block , sizeof(DTYPE)*threads_per_block>>>(d_vector,nvalues);
		nvalues = nblocks;
		nblocks = (nvalues + threads_per_block - 1 )/threads_per_block;
	}

    DTYPE *h_vector;
    h_vector = (DTYPE *) malloc(sizeof(DTYPE));

    //DTYPE *sum=0;
    DTYPE step = (1.0)/((DTYPE)nsteps);

    hipMemcpy(h_vector,d_vector,sizeof(DTYPE),hipMemcpyDeviceToHost);

    //
    
    printf("The values of Pi  is %f",*h_vector*step*4.0);


    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

   
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
    
    //cleanup memory
    free(h_vector);
    checkCudaErrors(hipFree(d_vector));

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
